

#include <hip/hip_runtime.h>
#include "hip/hip_complex.h"
#include <complex.h>


/*
* Add a constant to a vector.
*/
__device__ int modulo(int m, int n) { return m >= 0 ? m % n : (n - abs(m % n)) % n; }

__device__ __forceinline__ hipComplex cexpf(hipComplex z)
{

	hipComplex res;
	float t = expf(z.x);
	sincosf(z.y, &res.y, &res.x);
	res.x *= t;
	res.y *= t;
	return res;

}

__device__ float2  mat3_MULT_coeff(float2* MAT1, float2* MAT2, int i, int j) {
	float2 val = make_hipFloatComplex(0, 0);
	for (int k = 0; k < 3; k++) {
		val = hipCaddf(val, hipCmulf(MAT1[k * 3 + i], MAT2[j * 3 + k]));
	}
	return val;
}
__device__ float2  mat3_MULT_coeff(float2* MAT1, float* MAT2, int i, int j) {
	float2 val = make_hipFloatComplex(0, 0);
	for (int k = 0; k < 3; k++) {
		val = hipCaddf(val, hipCmulf(MAT1[k * 3 + i], make_hipFloatComplex(MAT2[j * 3 + k],0)));
	}
	return val;
}
__device__ float2  mat3_MULT_coeff(float* MAT1, float2* MAT2, int i, int j) {
	float2 val = make_hipFloatComplex(0, 0);
	for (int k = 0; k < 3; k++) {
		val = hipCaddf(val, hipCmulf(make_hipFloatComplex(MAT1[k * 3 + i],0), MAT2[j * 3 + k]));
	}
	return val;
}

struct convolve_params {
	int pos_A_shared;
	int pos_B_shared;
	int pos_Uz_shared;
	int pos_V0_shared;
	int pos_Vout_shared;
	int size_2D;
	int size_z;
	int size_angle;
};
__global__ void convolve_kernel(float const* const A_global, float const* const B_global, float const* const Uz_global, float2 const* const V0_global, float2* Vout_global, 
	int pos_A_shared,
	int pos_B_shared,
	int pos_Uz_shared,
	int pos_V0_shared,
	int pos_Vout_shared,
	int pos_Vp_shared,
	int pos_temp_shared,
	int pos_tmat_shared,
	int size_2D,
	int size_z,
	int size_angle,
	float res_z, 
	float start_z)
{
	extern __shared__ float data_shared[];

	int column_num = blockIdx.x; // the global column of data
	int vertical_index = threadIdx.x; //the vertical position of the data

	float* A = &data_shared[pos_A_shared];
	float* B = &data_shared[pos_B_shared];
	float* Uz = &data_shared[pos_Uz_shared];
	float2* V0 = (float2*)(&data_shared[pos_V0_shared]);
	float2* Vout = (float2*)(&data_shared[pos_Vout_shared]); 
	float2* Vp = (float2*)(&data_shared[pos_Vp_shared]);
	float2* temp = (float2*)(&data_shared[pos_temp_shared]);
	float2* tmat= (float2*)(&data_shared[pos_tmat_shared]);

	//load data 
	if (column_num < size_2D) {//verify we are working on a valid column
		
		int start_field = 0;
		while (start_field + vertical_index < size_angle * 9) {

			A[start_field + vertical_index] = A_global[9 * column_num + (start_field + vertical_index) % 9 + ((start_field + vertical_index) / 9) * size_2D * 9];
			B[start_field + vertical_index] = B_global[(start_field + vertical_index) % 9 + ((start_field + vertical_index) / 9) * 9];
			V0[start_field + vertical_index] = V0_global[9 * column_num + (start_field + vertical_index) % 9 + ((start_field + vertical_index) / 9) * size_2D * 9];

			start_field = start_field + size_z;
		}
		
		start_field = 0;
		while (start_field + vertical_index < size_angle) {
			Uz[start_field + vertical_index] = Uz_global[column_num + (start_field + vertical_index) * size_2D];

			start_field = start_field + size_z;
		}

		if (vertical_index < size_z) {
			for (int k = 0; k < 9; k++) {
				Vout[vertical_index + k * size_z] = make_hipFloatComplex(0, 0);
			}
		}
		start_field = 0;
		while (start_field + vertical_index < size_z * 9) {
			Vp[start_field + vertical_index]=Vout_global[9 * column_num + (start_field + vertical_index) % 9 + ((start_field + vertical_index) / 9) * size_2D * 9];
			start_field = start_field + size_z;
		}
		

		//summ
		for (int angle = 0; angle < size_angle; angle++) {

			__syncthreads();

			int vertical_skip = ((size_z) / 9);
			int mat_id = vertical_index % 9;
			int vert_id = vertical_index / 9;

			if (vertical_index < vertical_skip * 9) {

				temp[vertical_index] = make_hipFloatComplex(0, 0);



				//sum like some in parallel
				for (int k = 0; vert_id + k * vertical_skip < size_z; k++) {
					int curr_index = mat_id + (vert_id + k * vertical_skip) * 9;
					int z = (vert_id + k * vertical_skip);
					if (curr_index < size_z * 9) {
						float2 uz_val = make_hipFloatComplex(0,- 6.283185307179586f * (Uz[angle]) * (float(z) * res_z + start_z));
						temp[vertical_index] = hipCaddf(temp[vertical_index],
							hipCmulf(Vp[curr_index], cexpf(uz_val))
						);
					}
				}
			}
			__syncthreads();

			//sum the rest in series
			if (vert_id == 0) {
				for (int k = 1; k < vertical_skip; k++) {
					temp[vertical_index] = hipCaddf(temp[vertical_index], temp[vertical_index + k * 9]);
				}
				temp[vertical_index] = hipCsubf(temp[vertical_index], V0[angle * 9 + vertical_index]);
			}
			__syncthreads();
			if (vert_id == 0) {
				tmat[vertical_index] = mat3_MULT_coeff(&A[angle * 9], &temp[0], vertical_index % 3, vertical_index / 3);
			}
			__syncthreads();
			if (vert_id == 0) {
				temp[vertical_index] = mat3_MULT_coeff(&tmat[0], &B[angle * 9], vertical_index % 3, vertical_index / 3);
			}

			// resumation 

			__syncthreads();
			start_field = 0;
			while (start_field + vertical_index < size_z *9) {
				int z = (start_field + vertical_index)/9;
				float2 uz_val = make_hipFloatComplex(0, 6.283185307179586f * (Uz[angle]) * (float(z) * res_z + start_z));
				Vout[start_field + vertical_index] = hipCaddf(Vout[start_field + vertical_index], hipCmulf(temp[(start_field + vertical_index) % 9], cexpf(uz_val)));
				start_field = start_field + size_z;
			}
			
		}
		//save data 
		__syncthreads();
		start_field = 0;
		while (start_field + vertical_index < size_z * 9) {
			int mat_pos = (start_field + vertical_index) % 9;
			int mat_pos_trans = (mat_pos % 3) * 3 + (mat_pos / 3);
			float2 val_symetric = hipCaddf(Vout[start_field + vertical_index], Vout[start_field + vertical_index - mat_pos + mat_pos_trans]);
			//val_symetric = Vout[start_field + vertical_index];
			Vout_global[9 * column_num + (start_field + vertical_index) % 9 + ((start_field + vertical_index) / 9) * size_2D * 9] = hipCmulf(val_symetric, make_hipFloatComplex(0.5,0));
			
			start_field = start_field + size_z;
		}
	}
}
